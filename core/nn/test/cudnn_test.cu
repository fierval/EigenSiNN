#include <gtest/gtest.h>
#include "device/device_tensor.hpp"
#include <cudnn/cudnn_workspace.hpp>

#include "include/commondata4d.hpp"
#include "include/convdata4d.hpp"
#include "ops/comparisons.hpp"

using namespace Eigen;
using namespace EigenSinn;

namespace EigenSinnTest {
  class CudnnTest : public ::testing::Test {

  protected:
    void SetUp() override {
      cd.init();
      cd1p.init();

      params = std::make_shared<ConvolutionParams<4>>(cd.convInput.dimensions(), cd.convWeights.dimensions(), padding, stride, dilation, false, true);
      W = std::make_shared<CudnnWorkspace>(*params);
    }

    CommonData4d<GpuDevice, RowMajor> cd;
    ConvDataWith1Padding<GpuDevice, RowMajor> cd1p;

    const int stride = 1, dilation = 1;
    const Padding2D padding{ 0, 0 };

    std::shared_ptr<ConvolutionParams<4> >params;

    std::shared_ptr<CudnnWorkspace> W;
  };

  TEST_F(CudnnTest, SimpleConvForward) {

    //Create all the descriptors
    // - cudnn
    // - input tensor
    // - filter
    // - convolution
    DeviceTensor<float, 4, GpuDevice, RowMajor> out(params->output_dims());

    // forward convolution
    checkCudnnErrors(hipdnnConvolutionForward(CudnnWorkspace::cudnn(), &(CudnnWorkspace::one), W->input_desc, cd.convInput->data(),
      W->filter_desc, cd.convWeights->data(), W->conv_desc, W->conv_fwd_algo, CudnnWorkspace::workspace(), CudnnWorkspace::workspace_size,
      &(CudnnWorkspace::zero), W->output_desc, out->data()));

    EXPECT_TRUE(is_elementwise_approx_eq(cd.output, out));

  }

  TEST_F(CudnnTest, SimpleConvBackward) {

    //Create all the descriptors
    // - cudnn
    // - input tensor
    // - filter
    // - convolution

    DeviceTensor<float, 4, GpuDevice, RowMajor> out(params->output_dims());
    DeviceTensor<float, 4, GpuDevice, RowMajor> dinput(params->orig_dims());
    DeviceTensor<float, 4, GpuDevice, RowMajor> dweight(params->kernel_dims);

    // forward convolution
    checkCudnnErrors(hipdnnConvolutionForward(CudnnWorkspace::cudnn(), &(CudnnWorkspace::one), W->input_desc, cd.convInput->data(),
      W->filter_desc, cd.convWeights->data(), W->conv_desc, W->conv_fwd_algo, CudnnWorkspace::workspace(), CudnnWorkspace::workspace_size,
      &(CudnnWorkspace::zero), W->output_desc, out->data()));

    EXPECT_TRUE(is_elementwise_approx_eq(cd.output, out));

    // data backwards
    checkCudnnErrors(hipdnnConvolutionBackwardData(CudnnWorkspace::cudnn(), &(CudnnWorkspace::one), W->filter_desc, cd.convWeights->data(),
      W->output_desc, cd.convLoss->data(), W->conv_desc, W->conv_bwd_data_algo, CudnnWorkspace::workspace(), CudnnWorkspace::workspace_size, &(CudnnWorkspace::zero), W->input_desc, dinput->data()));


    EXPECT_TRUE(is_elementwise_approx_eq(cd.dinput, dinput));

    // weights backwards
    checkCudnnErrors(
      hipdnnConvolutionBackwardFilter(CudnnWorkspace::cudnn(), &(CudnnWorkspace::one), W->input_desc, cd.convInput->data(), W->output_desc, cd.convLoss->data(),
        W->conv_desc, W->conv_bwd_filter_algo, CudnnWorkspace::workspace(), CudnnWorkspace::workspace_size, &(CudnnWorkspace::zero), W->filter_desc, dweight->data()));

    EXPECT_TRUE(is_elementwise_approx_eq(cd.dweight, dweight));
  }
}